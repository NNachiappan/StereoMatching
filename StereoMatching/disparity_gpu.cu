#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include ""
#include <iostream>

#include <opencv2/core.hpp>

#include "disparity_gpu.h"

using namespace std;


//Compare two windows using SSD
__device__ unsigned int ssd(int leftIdx, int rightIdx, unsigned char* leftShared, unsigned char* rightShared, short windSize, int n) {

	unsigned int SSD = 0;

	for (int i = 0; i < windSize; i++) {
		for (int j = 0; j < windSize; j++) {

			int idx = j + (i * (n+windSize-1));
			int diff = leftShared[leftIdx + idx] - rightShared[rightIdx + idx];

			SSD += diff * diff;
		}
	}

	return SSD;
};


//Main kernel function
__global__ void disparity(unsigned short int* dispImage, unsigned char* left, unsigned char* right, int m, int n, short windSize, const int numCalcs, const int pixPBlock, unsigned int* disps) {


	//Shared memory declaration
	extern __shared__ unsigned char shared[];
	unsigned char *leftShared = shared;
	unsigned char *rightShared = &leftShared[pixPBlock];

	//load left and right image data into shared memory specific to each block
	for (int i = 0; i < ceilf((float)pixPBlock/blockDim.x); i++) {
		int idx = threadIdx.x + i * blockDim.x;
		if (idx < pixPBlock) {
			leftShared[idx] = left[idx + blockIdx.x * (n + windSize-1)];
			rightShared[idx] = right[idx + blockIdx.x * (n + windSize-1)];
		}
	}

	__syncthreads();

	//Threshold
	unsigned short int thresh = (int) (floorf(0.1 * n)+1);


	//Calculate all disparities within the threshold range.
	for (int i = 0; i < ceilf((float)numCalcs / blockDim.x); i++) {
		int idx = threadIdx.x + i * blockDim.x;
		
		if (idx < numCalcs) {
			//Get the index of the current pixel of the row in the left image and
			// the index of the pixel it is being compared to in the right image.
			int leftIdx = (int) floorf((float)idx/thresh);
			int rightIdx = idx - (leftIdx * thresh);

			int rightSSDIdx = fmaxf(rightIdx, leftIdx - (thresh - rightIdx));

			if (rightIdx <= leftIdx) {
				disps[(rightIdx*n) + leftIdx + numCalcs * blockIdx.x] = ssd(leftIdx, rightSSDIdx, leftShared, rightShared, windSize, n);
			}
		}
	}

	__syncthreads();


	//Find min ssd and copy disparity to global memory
	for (int i = 0; i < ceilf((float)n/blockDim.x); i++) {
		int idx = threadIdx.x + i * blockDim.x;

		if (idx < n) {
			unsigned int minSSD = 4294967295;
			unsigned short int minJ = 0;

			for (int j = 0; j < thresh; j++) {
				if (disps[idx + (j*n) + numCalcs * blockIdx.x] < minSSD) {
					minSSD = disps[idx + (j * n) + numCalcs * blockIdx.x];
					minJ = j;
				}
			}

			//copy disparity to global mem
			dispImage[idx + blockIdx.x * n] = (int) fminf(idx - minJ, thresh - minJ);
			
		}
	}

}

void disparityGPU(cv::Mat& dispImage, cv::Mat& left, cv::Mat& right, int m, int n, short windSize) {

	//Timer code
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);



	unsigned short int* d_disparity;
	unsigned char *d_left, *d_right;
	
	unsigned int* d_disps;
	

	const int numCalcs = n * (floor(n * 0.1)+1);
	const int pixPBlock = left.cols * windSize;

	unsigned int* disps = new unsigned int[numCalcs*m];
	fill(disps, disps + numCalcs * m, UINT_MAX);

	hipMalloc(&d_disparity, dispImage.step * dispImage.rows);
	hipMalloc(&d_left, left.step * left.rows);
	hipMalloc(&d_right, right.step * right.rows);
	hipMalloc(&d_disps, sizeof(unsigned int) * numCalcs * m);

	hipMemcpy(d_disparity, dispImage.ptr(), dispImage.step * dispImage.rows, hipMemcpyHostToDevice);
	hipMemcpy(d_left, left.ptr(), left.step * left.rows, hipMemcpyHostToDevice);
	hipMemcpy(d_right, right.ptr(), right.step * right.rows, hipMemcpyHostToDevice);
	hipMemcpy(d_disps, disps, sizeof(unsigned int) * numCalcs * m, hipMemcpyHostToDevice);

	hipEventRecord(start);
	disparity<<<m, 1024, (sizeof(unsigned char) * 2 * (left.cols) * windSize) >>>(d_disparity, d_left, d_right, m, n, windSize, numCalcs, pixPBlock, d_disps);
	hipEventRecord(stop);


	hipMemcpy(dispImage.ptr(), d_disparity, dispImage.step * dispImage.rows, hipMemcpyDeviceToHost);
	hipEventSynchronize(stop);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	cout << "GPU Time ms: " << milliseconds << endl;


	hipFree(d_disparity);
	hipFree(d_left);
	hipFree(d_right);
	hipFree(d_disps);

	delete[] disps;

}
